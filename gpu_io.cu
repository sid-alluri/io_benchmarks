#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <fstream>

using namespace std;
using namespace std::chrono;

const vector<size_t> DATA_SIZES = {10 * 1024 * 1024, 50 * 1024 * 1024, 100 * 1024 * 1024, 250 * 1024 * 1024, 500 * 1024 * 1024, 1024 * 1024 * 1024};

struct BenchmarkResult
{
    size_t dataSize;
    double h2dLatency; // Host to Device latency in milliseconds
    double d2hLatency; // Device to Host latency in milliseconds
};

void benchmarkMemoryTransfer(size_t dataSize, vector<BenchmarkResult> &results)
{
    // Allocate host memory
    vector<char> h_data(dataSize, 'a');
    char *d_data;

    // Allocate device memory
    hipMalloc(&d_data, dataSize);

    // Measure host to device transfer time
    auto start = high_resolution_clock::now();
    hipMemcpy(d_data, h_data.data(), dataSize, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    auto end = high_resolution_clock::now();
    auto elapsed = duration_cast<milliseconds>(end - start);
    double h2dLatency = elapsed.count();
    cout << "Host to Device Transfer Latency for " << dataSize / (1024 * 1024) << " MB: " << h2dLatency << " ms" << endl;

    // Measure device to host transfer time
    start = high_resolution_clock::now();
    hipMemcpy(h_data.data(), d_data, dataSize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();
    elapsed = duration_cast<milliseconds>(end - start);
    double d2hLatency = elapsed.count();
    cout << "Device to Host Transfer Latency for " << dataSize / (1024 * 1024) << " MB: " << d2hLatency << " ms" << endl;

    // Free device memory
    hipFree(d_data);

    // Store the result
    results.push_back({dataSize, h2dLatency, d2hLatency});
}

void writeResultsToCSV(const vector<BenchmarkResult> &results, const string &fileName)
{
    ofstream outfile(fileName);

    if (!outfile)
    {
        cerr << "Failed to open CSV file for writing: " << fileName << endl;
        return;
    }

    outfile << "Data Size (MB),Host to Device Latency (ms),Device to Host Latency (ms)" << endl;

    for (const auto &result : results)
    {
        outfile << result.dataSize / (1024 * 1024) << ","
                << result.h2dLatency << ","
                << result.d2hLatency << endl;
    }

    outfile.close();
}

int main()
{
    vector<BenchmarkResult> results;

    for (size_t dataSize : DATA_SIZES)
    {
        benchmarkMemoryTransfer(dataSize, results);
    }

    // Write results to CSV
    writeResultsToCSV(results, "cuda_memory_transfer_benchmark_results.csv");

    return 0;
}
